#include "hip/hip_runtime.h"
//Author: Adriel Kim
//6-27-2020
/*
Desc: Basic 2D matrix operations such as element-wise addition, subtraction, multiplication, and division.
In addition, slightly more complex operations such as dot product.

Challenge: Learning how to represent 2D matices in C++ and mapping 2D indices to kernel

*/

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>

#define N (100)//# of elements in matrices
using namespace std;

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);
//any advantages with mapping directly to strucutre of matrix? We're just representing 2D matrix using 1D array...
//it would be difficult to do the above since we want the operations to occur over abitrarily large matrices
//this can definitely be optimzied by elminating redundant calculations
__global__ void matrixAddKernel(int* c, const int* a, const int* b) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        //adds total number of running threads to tid, the current index.
        tid += blockDim.x * gridDim.x;
    }
}
__global__ void matrixSubtractKernel(int* c, const int* a, const int* b) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] - b[tid];
        //adds total number of running threads to tid, the current index.
        tid += blockDim.x * gridDim.x;
    }
}
__global__ void matrixMultiplyKernel(int* c, const int* a, const int* b) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

__global__ void matrixDivideKernel(int* c, const int* a, const int* b) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = (a[tid] / b[tid]);
        tid += blockDim.x * gridDim.x;
    }
}

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
